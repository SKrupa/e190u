// Simple CUDA example
 

#include <hip/hip_runtime.h>
#include <stdio.h>
 
const int N = 16; 
const int blocksize = 16; 
 
__global__ void hello(char *a, int *b) 
{
	a[threadIdx.x] += b[threadIdx.x];
}

// Kernel definition Vector Add
__global__ void VecAdd(float* A, float* B, float* C)
{
 int i = threadIdx.x;
 C[i] = A[i] + B[i];
}
 
int main()
{
	float a[N], b[N], c[N];

	// Initial values
	for (int i = 0; i < N; ++i)
	{
		a[i] = 0.1f*i*i;
		b[i] = 0.1f*i;
	}
 
 float *ad, *bd, *cd;
 const int size = N*sizeof(float);
 
 // print a, b, c before
	printf("a\tb\tc\n");
 	for (int i = 0; i < N; ++i)
	{
		printf("%.2f\t%.2f\t%.2f\n", a[i], b[i], c[i]);
	}
 
 hipMalloc( (void**)&ad, size ); 
 hipMalloc( (void**)&bd, size ); 
 hipMalloc( (void**)&cd, size ); 
 hipMemcpy( ad, a, size, hipMemcpyHostToDevice ); 
 hipMemcpy( bd, b, size, hipMemcpyHostToDevice ); 
 
 dim3 dimBlock( blocksize, 1 );
 dim3 dimGrid( 1, 1 );
 VecAdd<<<dimGrid, dimBlock>>>(ad, bd, cd);
 hipMemcpy( c, cd, size, hipMemcpyDeviceToHost ); 
 hipFree( ad );
 hipFree( bd );
 hipFree( cd );
 
 // print results
	printf("a\tb\tc\n");
 	for (int i = 0; i < N; ++i)
	{
		printf("%.2f\t%.2f\t%.2f\n", a[i], b[i], c[i]);
	}

 return EXIT_SUCCESS;
}

