// Simple CUDA example
 

#include <hip/hip_runtime.h>
#include <stdio.h>
 
const int N = 16; 
 
__global__ void hello(char *a, int *b) 
{
	a[threadIdx.x] += b[threadIdx.x];
}

// Kernel definition Vector Add
__global__ void VecAdd(float* A, float* B, float* C)
{
 int i = threadIdx.x;
 C[i] = A[i] + B[i];
}

// Kernel definition Matrix Add
__global__ void MatAdd(float *A, float *B, float *C, int N)
{
 int i = threadIdx.x;
 int j = threadIdx.y;
 if (i < N && j < N)
 {
	 C[i*N+j] = A[i*N+j] + B[i*N+j];
 }
}

void printMat(float *a, int N, const char* name)
{
	printf("%s\n",name);
 	for (int i = 0; i < N; ++i)
	{
		for (int j = 0; j < N; ++j)
		{
			printf("%.2f\t", a[i*N+j]);
		}
		printf("\n");
	}
}
 
int main()
{
//	float a[N][N], b[N][N], c[N][N];
	float *a, *b, *c;

	a = (float *)malloc(N*N*sizeof(float));
	b = (float *)malloc(N*N*sizeof(float));
	c = (float *)malloc(N*N*sizeof(float));

	// Initial values
	for (int i = 0; i < N; ++i)
	{
		for (int j = 0; j < N; ++j)
		{
			a[i*N+j] = 0.1f*i*j;
			b[i*N+j] = 0.1f*(i+j*N);
		}
	}
 
 // NxN now
 float *ad, *bd, *cd;
 const int size = N*N*sizeof(float);
 
 // print a, b, c before
 	printMat(a,N,"a");
 	printMat(b,N, "b");
 	printMat(c,N, "c before");
 
 hipMalloc( (void**)&ad, size ); 
 hipMalloc( (void**)&bd, size ); 
 hipMalloc( (void**)&cd, size ); 
 hipMemcpy( ad, a, size, hipMemcpyHostToDevice ); 
 hipMemcpy( bd, b, size, hipMemcpyHostToDevice ); 
 
 // NxN block now
 dim3 dimBlock( N, N );
 dim3 dimGrid( 1, 1 );
 MatAdd<<<dimGrid, dimBlock>>>(ad, bd, cd, N);
 hipMemcpy( c, cd, size, hipMemcpyDeviceToHost ); 
 hipFree( ad );
 hipFree( bd );
 hipFree( cd );
 
 // print results
 	printMat(a,N, "a");
 	printMat(b,N, "b");
 	printMat(c,N, "c after");

 return EXIT_SUCCESS;
}

